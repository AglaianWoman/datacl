#include <time.h>
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <string>

#include "filter.hpp"
#include "filter.cuh"
using namespace std;

#define NELEM 5e8
#define NDICT 5000

#define TIMETHIS(command, startTimer, endTimer)	\
gettimeofday(startTimer, NULL);			\
command;					\
gettimeofday(endTimer, NULL);			\



typedef long long myType;

/**
* @brief get time difference between timers star and end
*
* @param start start timer - has to be struct timeval pointer
* @param end end timer - has to be struct timeval pointer
*
* @return time difference in millisecond
*/
double getTimeDiff( struct timeval& start, struct timeval& end )
{
    return ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec)) / 1000 ;
}


/**
* @brief verifies if the given arrays are identical up to dataSize elements
*
* @param arrGPU[] GPU result
* @param arrCPU[] CPU result
* @param dataSize number of elements to compare
*/
template <typename T>
void verifyResult( T arrGPU[], T arrCPU[], size_t dataSize )
{ 
    for(long long i=0; i < dataSize; i++)
    {
            if(arrGPU[i] != arrCPU[i])
            {
                    cout << "Results dont match!!! at " << i << " gpu: " << arrGPU[i] << " cpu: " << arrCPU[i] << endl;
                    break;
            }
    }
}

/**
* @brief generates random data
*
* @tparam T template parameter
* @param datavec data array
* @param dataSize number of elements
*/
template <typename T>
void data_generate(T* datavec, size_t dataSize, T low, T high)
{
	for( long long i=0; i<dataSize; i++ )
		//generating between 0 and RAND_MAX
		datavec[i] = low + (high - low) * ((double) rand() / (double) RAND_MAX);
}





/**
* @brief testing function to test and time filter (serial and GPU version) with random inputs.
*/
void testfilter()
{
    // timers
    struct timeval start, end;

    // data array
    myType *h_data = (myType *) malloc(NELEM * sizeof(myType));
    // allocate mem for the gpu result on host side
    char *h_bitvec = (char *) malloc(NELEM * sizeof(char));
    // allocate mem for cpu result array for verification
    char *bitvec = (char *) malloc(NELEM * sizeof(char));

    // initalize the memory
    data_generate( h_data, NELEM, (myType)0, (myType)RAND_MAX );
    myType val = rand();

    // declare functor for comparator callback function
    opFunctor<myType> opFuncNew;
    int opcode = rand() % 5;//    op_t opNew = GT;
    opFuncNew.setOp(opcode);

    cout << "Checking for value " << val << " and opcode " << opcode << endl;

    // run on gpu
    TIMETHIS( filterGPU_wrap <myType> ( h_data, NELEM, val, h_bitvec, opFuncNew), &start, &end );

    cout << "Time Taken by GPU: " << getTimeDiff(start, end) << "ms" << endl;

    // run on host for comparison
    TIMETHIS( filter<myType> (h_data, (size_t)NELEM, val, bitvec, opFuncNew), &start, &end );

    cout << "Time Taken by CPU: " << getTimeDiff(start, end) << "ms" << endl;

    verifyResult<char>( h_bitvec, bitvec, NELEM );

    // free host memory
    free(h_data);
    free(h_bitvec);
    free(bitvec);
}



/**
* @brief testing function to test and time filter (serial and GPU version with pinned memory with hipHostRegister()) with random inputs.
*/
void testfilter_PinnedRegMem()
{
    // timers
    struct timeval start, end;

    // data array
    myType *h_data = (myType *) malloc(NELEM * sizeof(myType));
    // allocate mem for the gpu result (pinned mem) on host side
    char *h_bitvec_pin = (char *) malloc(NELEM * sizeof(char));
    // allocate mem for cpu result array for verification
    char *bitvec = (char *) malloc(NELEM * sizeof(char));


    // initalize the memory
    data_generate( h_data, NELEM, (myType)0, (myType)RAND_MAX );

    myType val = rand();

    // declare functor for comparator callback function
    opFunctor<myType> opFuncNew;
    int opcode = rand() % 5; //    op_t opNew = GT;
    opFuncNew.setOp(opcode);

    cout << "Checking for value " << val << " and opcode " << opcode << endl;

    // run on gpu
    TIMETHIS( filterGPU_wrap_regPin <myType> ( h_data, NELEM, val, h_bitvec_pin, opFuncNew), &start, &end );

    cout << "Time Taken by GPU (pinned mem using hipHostRegister): " << getTimeDiff(start, end) << "ms" << endl;




    // run on host for comparison
    TIMETHIS( filter<myType> (h_data, (size_t)NELEM, val, bitvec, opFuncNew), &start, &end );

    cout << "Time Taken by CPU: " << getTimeDiff(start, end) << "ms" << endl;

    verifyResult<char>( h_bitvec_pin, bitvec, NELEM );

    // free host memory
    free(h_data);
    free(h_bitvec_pin);
    free(bitvec);
}





/**
* @brief testing function to test and time filter (serial and GPU version with pinned memory with hipHostAlloc()) with random inputs.
*/
void testfilter_PinnedMem()
{
    // timers
    struct timeval start, end;

    myType *h_data_p;
    char* h_bitvec_p;

    checkCudaErrors(hipHostAlloc((void **)&h_data_p, NELEM * sizeof(myType), hipHostMallocMapped));
    // allocate mem for the gpu result on host side
    checkCudaErrors(hipHostAlloc((void **)&h_bitvec_p, NELEM * sizeof(char), hipHostMallocMapped));
    // allocate mem for cpu result array for verification
    char* bitvec = (char *) malloc(NELEM * sizeof(char));

    // initalize the memory
    data_generate( h_data_p, NELEM, (myType)0, (myType)RAND_MAX);

    myType val = rand();

    // declare functor for comparator callback function
    opFunctor<myType> opFuncNew;
    int opcode = rand() % 5;//    op_t opNew = GT;
    opFuncNew.setOp(opcode);

    cout << "Checking for value " << val << " and opcode " << opcode << endl;

    // run on gpu
    TIMETHIS( filterGPU_wrap_pin <myType> ( h_data_p, NELEM, val, h_bitvec_p, opFuncNew), &start, &end );

    cout << "Time Taken by GPU (pinned mem using hipHostAlloc()): " << getTimeDiff(start, end) << "ms" << endl;

    // run on host for comparison
    TIMETHIS( filter<myType> (h_data_p, (size_t)NELEM, val, bitvec, opFuncNew), &start, &end );


    verifyResult<char>( h_bitvec_p, bitvec, NELEM );

    cout << "Time Taken by CPU: " << getTimeDiff(start, end) << "ms" << endl;

    // free host memory
    free(bitvec);

    //free device memory
    checkCudaErrors(hipHostFree(h_data_p));
    checkCudaErrors(hipHostFree(h_bitvec_p));
}


/**
* @brief testing function for filter in.
*/
void testfilterIn()
{
    // timers
    struct timeval start, end;

    // data array
    myType *h_data = (myType *) malloc(NELEM * sizeof(myType));
    // data array
    myType *h_dict = (myType *) malloc(NDICT * sizeof(myType));
    // allocate mem for the gpu result on host side
    char *h_bitvec = (char *) malloc(NELEM * sizeof(char));
    // allocate mem for cpu result array for verification
    char *bitvec = (char *) malloc(NELEM * sizeof(char));

    // initalize the memory
    data_generate( h_data, NELEM, (myType)0, (myType)NELEM );

    // initalize the dictionary
    data_generate( h_dict, NDICT, (myType)0, (myType)NELEM );

    cout << "Checking for elements of input array in dictionary (using global memory of GPU to store the dictionary) " << endl;

    // run on gpu
    TIMETHIS( filterInGPU_wrap <myType> ( h_data, NELEM, h_dict, NDICT, h_bitvec ), &start, &end );

    cout << "Time Taken by GPU: " << getTimeDiff(start, end) << "ms" << endl;

    // run on host for comparison
    TIMETHIS( filterIn<myType> ( h_data, NELEM, h_dict, NDICT, bitvec ), &start, &end );

    cout << "Time Taken by CPU: " << getTimeDiff(start, end) << "ms" << endl;

    verifyResult<char>( h_bitvec, bitvec, NELEM );

    // free host memory
    free(h_data);
    free(h_bitvec);
    free(bitvec);
}


/**
* @brief testing function for filter in - this variant uses constant memory of the GPU which is a faster read-only memory.
*/
void testfilterIn_conMem()
{
    // timers
    struct timeval start, end;

    // data array
    myType *h_data = (myType *) malloc(NELEM * sizeof(myType));
    // data array
    myType *h_dict = (myType *) malloc(NDICT * sizeof(myType));

    // allocate mem for the gpu result on host side
    char *h_bitvec = (char *) malloc(NELEM * sizeof(char));
    // allocate mem for cpu result array for verification
    char *bitvec = (char *) malloc(NELEM * sizeof(char));

    // initalize the memory
    data_generate( h_data, NELEM, (myType)0, (myType)NELEM );

    // initalize the dictionary
    data_generate( h_dict, NDICT, (myType)0, (myType)NELEM );

    gettimeofday(&start, NULL);
    // allocate input and output data arrays
    myType *d_data;
    checkCudaErrors(hipMalloc( (void **) &d_data, NELEM * sizeof(myType) ));
    char *d_bitvec;
    checkCudaErrors(hipMalloc( (void **) &d_bitvec,  NELEM * sizeof(char) ));

    // copy data array to device
    checkCudaErrors(hipMemcpy(d_data, h_data, NELEM * sizeof(myType), hipMemcpyHostToDevice));

    cout << "Checking for elements of input array in dictionary (using constant memory of GPU to store the dictionary) " << endl;

    // run on gpu
    filterInGPU_conMem_wrap <myType> ( d_data, NELEM, h_dict, NDICT, d_bitvec );

    checkCudaErrors(hipMemcpy(h_bitvec, d_bitvec, NELEM * sizeof(char), hipMemcpyDeviceToHost));
    gettimeofday(&end, NULL);

    cout << "Time Taken by GPU: " << getTimeDiff(start, end) << "ms" << endl;

    // run on host for comparison
    TIMETHIS( filterIn<myType> ( h_data, NELEM, h_dict, NDICT, bitvec ), &start, &end );

    cout << "Time Taken by CPU: " << getTimeDiff(start, end) << "ms" << endl;

    verifyResult<char>( h_bitvec, bitvec, NELEM );

    // free device memory
    hipFree(d_data);
    hipFree(d_bitvec);

    hipDeviceReset();

    // free host memory
    free(h_data);
    free(h_bitvec);
    free(bitvec);
}




int main(int argc, char **argv)
{


srand( time(NULL) );
/*
cout << "-----------------------------------------------------" << endl;

testfilter();

cout << endl;
cout << endl;
cout << "-----------------------------------------------------" << endl;

testfilter_PinnedRegMem();


cout << endl;
cout << endl;
cout << "-----------------------------------------------------" << endl;

testfilter_PinnedMem();

cout << endl;
cout << endl;
cout << "-----------------------------------------------------" << endl;
*/

cout << "-----------------------------------------------------" << endl;
cout << endl;

testfilterIn();

cout << endl;
cout << "-----------------------------------------------------" << endl;


cout << "-----------------------------------------------------" << endl;
cout << endl;

testfilterIn_conMem();

cout << endl;
cout << "-----------------------------------------------------" << endl;

    return 0;
}

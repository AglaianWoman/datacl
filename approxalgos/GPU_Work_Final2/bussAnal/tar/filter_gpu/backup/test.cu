#include "hip/hip_runtime.h"
#include <time.h>
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#include <string>

//#include <bitset>
#include "filter.h"
using namespace std;

#define N 100000000
#define NCHAR 20
#define NTHREADS 512
#define NBLOCKS 512

enum op_t {EQ, LT, LE, GT, GE, NE};

typedef long long myType;

template <typename T>
struct opFunctor{
        private:
                int op;
        public:
		opFunctor() { op = EQ; }
		opFunctor(int opIn) { op = opIn; }
		__host__ __device__ void setOp (int opIn) { op = opIn; }
		__host__ __device__ bool operator() (const T &a, const T &b)
		{
			if( op == EQ ) //eq
				return (a == b);
			else if( op == LT ) //lt
				return (a < b);
			else if( op == LE ) //"le") )
				return (a <= b);
			else if( op == GT ) //"gt") )
				return (a > b);
			else if( op == GE ) //"ge") )
				return (a >= b);
			else if( op == NE ) //"ne") )
				return (a != b);
			else
				//by default, check for equal
				return (a == b);
		}
};

//Functors for 2-obj comparison
template <typename T>
struct opGreater{
        public:
		__host__ __device__ bool operator() (const T &a, const T &b)
		{ return (a > b); }
};

template <typename T>
struct opGreaterEqual{
        public:
		__host__ __device__ bool operator() (const T &a, const T &b)
		{ return (a >= b); }
};

template <typename T>
struct opLesser{
        public:
		__host__ __device__ bool operator() (const T &a, const T &b)
		{ return (a < b); }
};

template <typename T>
struct opLesserEqual{
        public:
		__host__ __device__ bool operator() (const T &a, const T &b)
		{ return (a <= b); }
};

template <typename T>
struct opNotEqual{
        public:
		__host__ __device__ bool operator() (const T &a, const T &b)
		{ return (a != b); }
};

template <typename T>
struct opEqual{
        public:
		__host__ __device__ bool operator() (const T &a, const T &b)
		{ return (a == b); }
};


//Filter GPU Kernel
template <typename T, class Compare> 
__global__ void filterGPU(T *data, size_t dataSize, const T val, char *bitvec, Compare comp) 
{
	//1-D thread index
        long long i = blockDim.x * blockIdx.x + threadIdx.x; 
 
	while(i < dataSize)
	{
		if( comp(data[i], val) ) 
			bitvec[i] = '1'; 
		else 
			bitvec[i] = '0'; 
		i += blockDim.x * gridDim.x;
	}
} 


int main(int argc, char **argv)
{
    struct timeval start, end;
    hipDeviceProp_t deviceProp;

    // This will pick the best possible CUDA capable device
    int devID = findCudaDevice(argc, (const char **)argv);
    
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
    

    myType *h_data = (myType *) malloc(N * sizeof(myType));

    // initalize the memory
    for (long long i = 0; i < N; ++i)
    {
	    h_data[i] = i;
    }

    myType val = 1000;

    // start timer
    gettimeofday(&start, NULL);

    // allocate device memory
    myType *d_data;
    checkCudaErrors(hipMalloc((void **) &d_data, N * sizeof(myType) ));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_data, h_data, N * sizeof(myType), hipMemcpyHostToDevice));


    // allocate device memory for result
    char *d_bitvec;
    checkCudaErrors(hipMalloc((void **) &d_bitvec, N * sizeof(char)));
    // allocate mem for the result on host side
    char *h_bitvec = (char *) malloc(N * sizeof(char));


    // Kernel configuration, where a one-dimensional
    // grid and one-dimensional blocks are configured.
    dim3 dimGrid(NBLOCKS);
    dim3 dimBlock(NTHREADS);

    // declare functor for comparator callback function
    opFunctor<myType> opFuncNew;
    op_t opNew = GT;
    opFuncNew.setOp(opNew);

    // execute the kernel
    filterGPU<<< dimGrid, dimBlock >>>(d_data, N, val, d_bitvec, opFuncNew);

    hipDeviceSynchronize();

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_bitvec, d_bitvec, N * sizeof(char), hipMemcpyDeviceToHost));

    gettimeofday(&end, NULL);
    double timeDiff_us = ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec));
    cout << "Time Taken by GPU: " << timeDiff_us / 1000 << "ms" << endl;

    // free device memory
    hipFree(d_data);
    hipFree(d_bitvec);

    hipDeviceReset();


    // result array for verification
    char *bitvec = (char *) malloc(N * sizeof(char));

    gettimeofday(&start, NULL);
    // run host code for comparison
    filter<myType> (h_data, (size_t)N, val, bitvec, opFuncNew);
    gettimeofday(&end, NULL);

    for(long long i=0; i < N; i++)
    {
	    if(bitvec[i] != h_bitvec[i])
	    {
		    cout << "Results dont match!!! at " << i << " gpu: " << bitvec[i] << " cpu: " << h_bitvec[i] << endl;
		    break;
	    }
    }

    timeDiff_us = ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec));
    cout << "Time Taken by CPU: " << timeDiff_us / 1000 << "ms" << endl;

    // free host memory
    free(h_data);
    free(h_bitvec);
    free(bitvec);

    return 0;

}

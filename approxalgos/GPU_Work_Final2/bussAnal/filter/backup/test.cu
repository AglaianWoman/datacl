#include "hip/hip_runtime.h"
#include <time.h>
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#include <string>

//#include <bitset>
#include "filter.h"
using namespace std;

#define N 1000000

typedef long long myType;

/*
template <class T>
struct opFunctor{
        private:
                int op;
        public:
		opFunctor() { op = "eq"; }
		opFunctor(string opIn) { op = opIn; }
		__device__ bool operator() (const T &a, const T &b)
		{
			if( op == "lt" )
				return (a < b);
			else if( op == "le" )
				return (a <= b);
			else if( op == "gt")
				return (a > b);
			else if( op == "ge")
				return (a >= b);
			else if( op == "ne")
				return (a != b);
			else
				//by default, check for equal
				return (a == b);
		}
};
*/

template <class T>
struct opGreater{
        public:
		__host__ __device__ bool operator() (const T &a, const T &b)
		{ return (a > b); }
};

template <class T, class Compare> 
__global__ void filterKernel(T *data, size_t dataSize, const T val, char *bitvec, Compare comp) 
{ 
 
        long long i = blockDim.x * blockIdx.x + threadIdx.x; 
 
        if(i < dataSize) 
        { 
                if( comp(data[i], val) ) 
                        bitvec[i] = '1'; 
                else 
                        bitvec[i] = '0'; 
        } 
} 


int main(int argc, char **argv)
{
	srand(time(NULL));
	struct timeval start, end;


//	myType data[N];
//	bitset<N> bitvec;
//	char bitvec[N];

//	for(long long i=0; i<N; i++)
//		data[i] = rand() % N;

//	myType val = rand() % N;


//    hipError_t error;
    hipDeviceProp_t deviceProp;

    // This will pick the best possible CUDA capable device
    int devID = findCudaDevice(argc, (const char **)argv);
    
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
    

    myType *h_data = (myType *) malloc(N * sizeof(myType));

    // initalize the memory
    for (long long i = 0; i < N; ++i)
    {
	    h_data[i] = i;
    }

    myType val = 1000;

    gettimeofday(&start, NULL);
    // allocate device memory
    myType *d_data;
    checkCudaErrors(hipMalloc((void **) &d_data, N * sizeof(myType) ));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_data, h_data, N * sizeof(myType), hipMemcpyHostToDevice));



    // allocate device memory for result
    char *d_bitvec;
    checkCudaErrors(hipMalloc((void **) &d_bitvec, N * sizeof(char)));
    // allocate mem for the result on host side
    char *h_bitvec = (char *) malloc(N * sizeof(char));



    // Kernel configuration, where a one-dimensional
    // grid and one-dimensional blocks are configured.
    long long Nthreads = 1024;
    long long Nblocks = (N + Nthreads - 1 ) / Nthreads;
    dim3 dimGrid(Nblocks);
    dim3 dimBlock(Nthreads);

    //opFunctor<myType> opNew;

    // execute the kernel
    filterKernel<<< dimGrid, dimBlock >>>(d_data, N, val, d_bitvec, opGreater<myType>());

    hipDeviceSynchronize();

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_bitvec, d_bitvec, N * sizeof(char), hipMemcpyDeviceToHost));

    hipFree(d_data);
    hipFree(d_bitvec);
    gettimeofday(&end, NULL);
    double timeDiff_us = ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec));
    cout << "Time Taken by GPU: " << timeDiff_us / 1000 << "ms" << endl;

    hipDeviceReset();






/*
//Available comparator object functions (functors)
equal_to
not_equal_to
greater
greater_equal
less
less_equal
*/




    char *bitvec = (char *) malloc(N * sizeof(char));

    gettimeofday(&start, NULL);
    filter<myType> (h_data, (size_t)N, val, bitvec, greater<myType> ());
    gettimeofday(&end, NULL);

    for(long long i=0; i < N; i++)
    {
	    if(bitvec[i] != h_bitvec[i])
	    {
		    cout << "Results dont match!!! at" << i << endl;
		    break;
	    }
    }

    timeDiff_us = ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec));
    cout << "Time Taken by CPU: " << timeDiff_us / 1000 << "ms" << endl;

//Print results to check if they are correct.
//	for(long long i=0; i<N; i++)
//		cout << "(" << data[i] << " " << bitvec[i] << ")" << endl;
//	cout << endl << "Value = " << val << endl;

return 0;

}

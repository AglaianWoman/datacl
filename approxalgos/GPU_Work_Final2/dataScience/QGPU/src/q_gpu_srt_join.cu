#include "hip/hip_runtime.h"
#include <iostream>
#include <map>
#include <limits>
#include <boost/shared_ptr.hpp>
#include "q_types.hpp"
#include "q_utilities.hpp"
#include "q_gpu_registers.hpp"
#include "q_gpu_server_kernels.hpp"
#include "q_gpu_srt_join.hpp"

using std::string;
using std::map;
using std::cerr;
using std::endl;
using std::numeric_limits;

using boost::shared_ptr;

namespace QDB {

  struct __srt_join_interface {
    __srt_join_interface () {
    }

    virtual ~__srt_join_interface () {
    }

    virtual int operator()(gpu_register const & RsrcLink,
                           gpu_register const & RsrcVal,
                           gpu_register const & RdstLink,
                           gpu_register & RdstVal,
                           gpu_register & RdstNn) {
      cerr << "ERROR: NOT IMPLEMENTED" << endl;
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
  };

  template <typename T_LINK,
            typename T_VALUE,
            typename MERGE_OP>
  struct __srt_join : public __srt_join_interface {
    __srt_join(T_VALUE init = 0)
      : __srt_join_interface(),
        init_val(init),
        notfound_val(init),
        notfound_nn_val(0) {
    }

    __srt_join(T_VALUE init, T_VALUE notfound, Q_I1_t notfound_nn)
      : __srt_join_interface(),
        init_val(init),
        notfound_val(notfound),
        notfound_nn_val(notfound_nn) {
    }

    virtual ~__srt_join() {
    }

    virtual int operator()(gpu_register const & RsrcLink,
                           gpu_register const & RsrcVal,
                           gpu_register const & RdstLink,
                           gpu_register & RdstVal,
                           gpu_register & RdstNn) {
      dim3 dimGrid(NBLOCKS);
      dim3 dimBlock(NTHREADS);
      srt_join_kernel<T_LINK, T_VALUE, MERGE_OP><<< dimGrid, dimBlock >>>
        (static_cast<T_LINK const *>(RsrcLink.reg),
         static_cast<T_VALUE const *>(RsrcVal.reg),
         size_t(RsrcLink.nR),
         static_cast<T_LINK const *>(RdstLink.reg),
         static_cast<T_VALUE *>(RdstVal.reg),
         static_cast<Q_I1_t *>(RdstNn.reg),
         size_t(RdstLink.nR),
         MERGE_OP(),
         init_val,
         notfound_val,
         notfound_nn_val);
      return Q_PASS;
    }
    T_VALUE init_val;
    T_VALUE notfound_val;
    Q_I1_t notfound_nn_val;
  };

  struct srt_join {
    srt_join() {
      add_link_ops<I1>();
      add_link_ops<I2>();
      add_link_ops<I4>();
      add_link_ops<I8>();
    }

    int operator()(gpu_register const & RsrcLink,
                   gpu_register const & RsrcVal,
                   gpu_register const & RdstLink,
                   std::string const & str_op,
                   gpu_register & RdstVal,
                   gpu_register & RdstNn) {
      if ((op.find(str_op) == op.end()) ||
          (op[str_op].find(RsrcLink.fldtype) == op[str_op].end()) ||
          (op[str_op][RsrcLink.fldtype].find(RsrcVal.fldtype) == op[str_op][RsrcLink.fldtype].end())) {
        cerr << "ERROR: srt_join not implemented for "
             << "type(link)=" << get_fldtype_string(RsrcLink.fldtype) << " "
             << "type(value)=" << get_fldtype_string(RsrcVal.fldtype) << " "
             << "op=" << str_op << endl;
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }

      return op[str_op][RsrcLink.fldtype][RsrcVal.fldtype]->operator()(RsrcLink, RsrcVal, RdstLink,
                                                                       RdstVal, RdstNn);
    }

  private:
    template <typename T>
    inline
    void addop(string const & opname, FLD_TYPE link_type, FLD_TYPE val_type,
               T * oper) {
      op[opname][link_type][val_type] = shared_ptr<__srt_join_interface>
        (static_cast<__srt_join_interface *>(oper));
    }

    template <FLD_TYPE link, FLD_TYPE value>
    inline
    void addops() {
      typedef typename fldtype_traits<link>::q_type link_type;
      typedef typename fldtype_traits<value>::q_type value_type;
      typedef typename fldtype_traits<value>::unsigned_type unsigned_value_type;
      addop("reg", link, value, new __srt_join<link_type, value_type, SECOND<value_type> >);
      addop("min", link, value, new __srt_join<link_type, value_type, MIN<value_type> >(numeric_limits<value_type>::max()));
      addop("max", link, value, new __srt_join<link_type, value_type, MAX<value_type> >(numeric_limits<value_type>::min()));
      addop("sum", link, value, new __srt_join<link_type, value_type, ADD<value_type> >(0, 0, 1));
      addop("and", link, value, new __srt_join<link_type, value_type, BWISE_AND<value_type> >(value_type(~unsigned_value_type(0)), 0, 1));
      addop("or", link, value, new __srt_join<link_type, value_type, BWISE_OR<value_type> >(0, 0, 1));
      addop("cnt", link, value, new __srt_join<link_type, value_type, FIRST_PLUS_ONE<value_type> >(0, 0, 1));
    }

    template <FLD_TYPE link>
    inline
    void add_link_ops() {
      addops<link, I1>();
      addops<link, I2>();
      addops<link, I4>();
      addops<link, I8>();
    }

    typedef map<FLD_TYPE, shared_ptr<__srt_join_interface> > map_valuefld_to_oper;
    typedef map<FLD_TYPE,  map_valuefld_to_oper> map_linkfld_to_valuefld;
    typedef map<string,  map_linkfld_to_valuefld> map_opname_to_linkfld;

    map_opname_to_linkfld op;
  };

  static srt_join s_srt_join;

  int srt_join_GPU(gpu_register const & RsrcLink,
                   gpu_register const & RsrcVal,
                   gpu_register const & RdstLink,
                   std::string const & str_op,
                   gpu_register & RdstVal,
                   gpu_register & RdstNn) {
    return s_srt_join(RsrcLink, RsrcVal, RdstLink, str_op, RdstVal, RdstNn);
  }

}

#include <thrust/logical.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <string>
#include <iostream>
#include <map>
#include <boost/shared_ptr.hpp>

#include "q_types.hpp"
#include "q_gpu_registers.hpp"
#include "q_gpu_server_kernels.hpp"
#include "q_gpu_count.hpp"

using std::string;
using std::cerr;
using std::endl;
using std::map;

using boost::shared_ptr;

namespace QDB {

  struct __are_all_indices_valid_interface {
    __are_all_indices_valid_interface() {
    }

    virtual ~__are_all_indices_valid_interface() {
    }

    virtual int operator()(gpu_register const & R, size_t ub, bool & result) {
      cerr << "ERROR: NOT IMPLEMENTED" << endl;
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
  };

  template <typename T>
  struct __are_all_indices_valid : public __are_all_indices_valid_interface {
    __are_all_indices_valid()
      : __are_all_indices_valid_interface() {
    }

    virtual ~__are_all_indices_valid() {
    }

    virtual int operator()(gpu_register const & R, size_t ub, bool & result) {
      int * d_val;
      int h_val = 0;
      if (make_d_copy(&d_val, h_val) != Q_PASS) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      dim3 dimGrid(NBLOCKS);
      dim3 dimBlock(NTHREADS);
      is_any_op_kernel<T, INVALID_INDEX<T> >
        <<< dimGrid, dimBlock >>>
        ((T *)(R.reg), size_t(R.nR), INVALID_INDEX<T>(ub), d_val);
      if (restore_h_copy(h_val, &d_val) != Q_PASS) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      result = (h_val == 0);
      return Q_PASS;
    }
  };

  class are_all_indices_valid {
  public:
    are_all_indices_valid() {
      addop<I1>();
      addop<I2>();
      addop<I4>();
      addop<I8>();
    }

    int operator()(gpu_register const & R, size_t ub, bool & result) {
      if (op.find(R.fldtype) == op.end()) {
        cerr << "ERROR: field type " << get_fldtype_string(R.fldtype) << " not supported for count or countf" << endl;
        result = false;
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      return (op[R.fldtype]->operator()(R, ub, result) != Q_PASS);
    }

  private:

    template <FLD_TYPE index>
    inline
    void addop() {
      typedef typename fldtype_traits<index>::q_type q_type;
      op[index] = shared_ptr<__are_all_indices_valid_interface>
        (static_cast<__are_all_indices_valid_interface *>
         (new __are_all_indices_valid<q_type>));
    }
    map<FLD_TYPE, shared_ptr<__are_all_indices_valid_interface> > op;
  };

  static are_all_indices_valid s_are_all_indices_valid;

  struct __count_interface {
    __count_interface () {
    }

    virtual ~__count_interface () {
    }

    virtual int operator()(gpu_register const & Ridx,
                           gpu_register const * pRcond,
                           gpu_register & Rdst) {
      cerr << "ERROR: NOT IMPLEMENTED" << endl;
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
  };

  template <typename T_INDEX, typename T_RESULT>
  struct __count : public __count_interface {
    __count()
      : __count_interface() {
    }

    virtual ~__count() {
    }

    virtual int operator()(gpu_register const & Ridx,
                           gpu_register const * pRcond,
                           gpu_register & Rdst) {
      if (hipMemset(Rdst.reg, 0, Rdst.nR * sizeof(T_RESULT)) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }

      dim3 dimGrid(NBLOCKS);
      dim3 dimBlock(NTHREADS);

      size_t sharedMemSize = Rdst.nR * sizeof(T_RESULT);
      if (sharedMemSize > MAX_SHARED_MEM) {
        sharedMemSize = 0;
      }
      count_kernel<T_INDEX, Q_I1_t, T_RESULT>
        <<< dimGrid, dimBlock, sharedMemSize >>>
        ((T_INDEX *)(Ridx.reg),
         pRcond ? (Q_I1_t const *)(pRcond->reg) : NULL,
         NULL,
         (T_RESULT *)(Rdst.reg),
         Ridx.nR, Rdst.nR, sharedMemSize);

      return Q_PASS;
    }
  };

  struct count {
  public:
    count() {
      add_index_ops<I1>();
      add_index_ops<I2>();
      add_index_ops<I4>();
    }

    int operator()(gpu_register const & Ridx,
                   gpu_register const * pRcond,
                   gpu_register & Rdst,
                   bool bSafeMode) {
      if (bSafeMode) {
        bool allgood = true;
        if ((s_are_all_indices_valid(Ridx, Rdst.nR, allgood) != Q_PASS) ||
            !allgood) {
          cerr << "ERROR: index out of valid range [0.." << Rdst.nR << ")" << endl;
          whereami(__FILE__, __LINE__);
          return Q_FAIL;
        }
      }

      if ((op.find(Ridx.fldtype) == op.end()) ||
          (op[Ridx.fldtype].find(Rdst.fldtype) == op[Ridx.fldtype].end()) ||
          (pRcond && pRcond->fldtype != I1)) {
        cerr << "ERROR: count not supported for "
             << "type(fs)=" << get_fldtype_string(Ridx.fldtype) << " ";
        if (pRcond) {
          cerr << "type(fc)=" << get_fldtype_string(pRcond->fldtype) << " ";
        }
        cerr << "type(fd)=" << get_fldtype_string(Rdst.fldtype) << endl;
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }

      return op[Ridx.fldtype][Rdst.fldtype]->operator()(Ridx, pRcond, Rdst);
    }

  private:

    template <FLD_TYPE index, FLD_TYPE result>
    inline
    void addop() {
      typedef typename fldtype_traits<index>::q_type index_type;
      typedef typename fldtype_traits<result>::q_type result_type;
      op[index][result] = shared_ptr<__count_interface>
        (static_cast<__count_interface *>
         (new __count<index_type, result_type>));
    }

    template <FLD_TYPE index>
    inline
    void add_index_ops() {
      addop<index, I4>();
      addop<index, I8>();
    }

    typedef map<FLD_TYPE, shared_ptr<__count_interface> > map_resultfld_to_oper;
    typedef map<FLD_TYPE, map_resultfld_to_oper> map_indexfld_to_resultfld;
    map_indexfld_to_resultfld op;
  };

  static count s_count;

  struct __countf_interface {
    __countf_interface() {
    }

    virtual ~__countf_interface() {
    }

    virtual int operator()(gpu_register const & Ridx,
                           gpu_register const * pRcond,
                           gpu_register const & Rval,
                           gpu_register & Rdst) {
      cerr << "ERROR: NOT IMPLEMENTED" << endl;
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
  };

  template <typename T_INDEX, typename T_VALUE, typename T_RESULT>
  struct __countf : public __countf_interface {
    __countf()
      :__countf_interface() {
    }

    virtual ~__countf() {
    }

    virtual int operator()(gpu_register const & Ridx,
                           gpu_register const * pRcond,
                           gpu_register const & Rval,
                           gpu_register & Rdst) {
      if (hipMemset(Rdst.reg, 0, Rdst.nR * sizeof(T_RESULT)) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }

      dim3 dimGrid(NBLOCKS);
      dim3 dimBlock(NTHREADS);
      size_t sharedMemSize = Rdst.nR * sizeof(T_RESULT);
      if (sharedMemSize > MAX_SHARED_MEM) {
        sharedMemSize = 0;
      }
      count_kernel<T_INDEX, T_VALUE, T_RESULT>
        <<< dimGrid, dimBlock, sharedMemSize >>>
        ((T_INDEX const *)(Ridx.reg),
         pRcond ? (Q_I1_t const *)(pRcond->reg) : NULL,
         (T_VALUE const *)(Rval.reg),
         (T_RESULT *)(Rdst.reg),
         Ridx.nR, Rdst.nR, sharedMemSize);
      return Q_PASS;
    }
  };

  struct countf {
    countf() {
      add_index_ops<I1>();
      add_index_ops<I2>();
      add_index_ops<I4>();
    }

    int operator()(gpu_register const & Ridx,
                   gpu_register const * pRcond,
                   gpu_register const & Rval,
                   gpu_register & Rdst,
                   bool bSafeMode) {
      if (bSafeMode) {
        bool allgood = true;
        if ((s_are_all_indices_valid(Ridx, Rdst.nR, allgood) != Q_PASS) ||
            !allgood) {
          cerr << "ERROR: index out of valid range [0.." << Rdst.nR << ")" << endl;
          whereami(__FILE__, __LINE__);
          return Q_FAIL;
        }
      }

      if ((op.find(Ridx.fldtype) == op.end()) ||
          (op[Ridx.fldtype].find(Rval.fldtype) == op[Ridx.fldtype].end()) ||
          (op[Ridx.fldtype][Rval.fldtype].find(Rdst.fldtype) == op[Ridx.fldtype][Rval.fldtype].end()) ||
          (pRcond && (pRcond->fldtype != I1))) {
        cerr << "ERROR: countf not supported for "
             << "type(fs)=" << get_fldtype_string(Ridx.fldtype) << " "
             << "type(fv)=" << get_fldtype_string(Rval.fldtype) << " ";
        if (pRcond) {
          cerr << "type(fc)=" << get_fldtype_string(pRcond->fldtype) << " ";
        }
        cerr << "type(fd)=" << get_fldtype_string(Rdst.fldtype) << endl;
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }

      return op[Ridx.fldtype][Rval.fldtype][Rdst.fldtype]->operator()(Ridx, pRcond, Rval, Rdst);
    }

  private:
    template <FLD_TYPE index, FLD_TYPE value, FLD_TYPE result>
    inline
    void addop() {
      typedef typename fldtype_traits<index>::q_type index_type;
      typedef typename fldtype_traits<value>::q_type value_type;
      typedef typename fldtype_traits<result>::q_type result_type;
      op[index][value][result] =
        shared_ptr<__countf_interface>
        (static_cast<__countf_interface *>
         (new __countf<index_type, value_type, result_type>));
    }

    template <FLD_TYPE index>
    inline
    void add_index_ops() {
      addop<index, I1, I4>();
      addop<index, I2, I4>();
      addop<index, I4, I4>();
      // addop<index, I8, I4>(); not supported since we do not want to sum I8s into an I4 result
      addop<index, I1, I8>();
      addop<index, I2, I8>();
      addop<index, I4, I8>();
      addop<index, I8, I8>();

      addop<index, F4, F4>();
      addop<index, F8, F8>();
    }

    typedef map<FLD_TYPE, shared_ptr<__countf_interface> > map_resultfld_to_oper;
    typedef map<FLD_TYPE, map_resultfld_to_oper> map_valuefld_to_resultfld;
    typedef map<FLD_TYPE, map_valuefld_to_resultfld> map_indexfld_to_valuefld;
    map_indexfld_to_valuefld op;
  };

  static countf s_countf;

  int
  count_GPU(gpu_register const & Ridx,
            gpu_register const * pRcond,
            gpu_register & Rdst,
            bool bSafeMode) {
    return s_count(Ridx, pRcond, Rdst, bSafeMode);
  }

  int
  countf_GPU(gpu_register const & Ridx,
             gpu_register const * pRcond,
             gpu_register const & Rval,
             gpu_register & Rdst,
             bool bSafeMode) {
    return s_countf(Ridx, pRcond, Rval, Rdst, bSafeMode);
  }

}

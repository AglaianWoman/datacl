#include "hip/hip_runtime.h"
#include "q_gpu_f1_shift_f2.hpp"
#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <iostream>
#include <map>
#include <limits>
#include <memory>
#include <boost/shared_ptr.hpp>
#include "q_types.hpp"
#include "q_gpu_registers.hpp"
#include "q_utilities.hpp"
#include "q_gpu_server_kernels.hpp"

using std::numeric_limits;
using std::cerr;
using std::endl;
using std::map;

using boost::shared_ptr;

namespace QDB {

  template <typename T>
  static inline int
  fill_device_memory(T * start, size_t sz, T value) {
    if (value) {
      boost::shared_ptr<T> h_start((T *)(malloc(sz * sizeof(T))), std::ptr_fun(free));
      if (!h_start.get()) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      for (T * i = h_start.get(), * iEnd = h_start.get() + sz; i != iEnd; ++i) {
        *i = value;
      }
      if (hipMemcpy((char *)start, (char const *)(h_start.get()),
                     sz * sizeof(T),hipMemcpyHostToDevice) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }

      /*
      // TODO: figure out why the thrust version below is slower than the above.
      thrust::fill(thrust::device_ptr<T>(start),
      thrust::device_ptr<T>(start + sz),
      value);
      */
    } else {
      if (hipMemset((char *)start, 0, sz * sizeof(T)) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
    }
    return Q_PASS;
  }

  template <typename T>
  static inline int
  copy_device_memory(T * dst, T const * src, size_t sz) {
    return ((hipMemcpy((char *)dst, (char const *)src, sz * sizeof(T), hipMemcpyDeviceToDevice) != hipSuccess) ? Q_FAIL : Q_PASS);
  }

  struct __f1_shift_f2_interface {
    __f1_shift_f2_interface() {
    }

    virtual ~__f1_shift_f2_interface() {
    }

    virtual int operator()(gpu_register const & Rin,
                           gpu_register & Rout,
                           int n_shift,
                           Q_I8_t fill_value) {
      cerr << "ERROR: NOT IMPLEMENTED" << endl;
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
  };

  template <typename T>
  struct __f1_shift_f2 : public __f1_shift_f2_interface {
    __f1_shift_f2()
      : __f1_shift_f2_interface() {
    }

    virtual ~__f1_shift_f2() {
    }

    virtual int operator()(gpu_register const & Rin,
                           gpu_register & Rout,
                           int n_shift,
                           Q_I8_t fill_value) {
      if (fill_value < Q_I8_t(numeric_limits<T>::min()) ||
          fill_value > Q_I8_t(numeric_limits<T>::max())) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      T value = T(fill_value);
      if (n_shift == 0) {
        if (copy_device_memory<T>((T *)(Rout.reg), (T const *)(Rin.reg), Rin.nR) != Q_PASS) {
          whereami(__FILE__, __LINE__);
          return Q_FAIL;
        }
      } else if ((abs(n_shift) >= Rin.nR)) {
        if (fill_device_memory<T>((T *)(Rout.reg), Rout.nR, value) != Q_PASS) {
          whereami(__FILE__, __LINE__);
          return Q_FAIL;
        }
      } else {
        if (n_shift > 0) {
          if (fill_device_memory<T>((T *)(Rout.reg), n_shift, value) != Q_PASS) {
            whereami(__FILE__, __LINE__);
            return Q_FAIL;
          }
          if (copy_device_memory<T>((T *)(Rout.reg) + n_shift, (T const *)(Rin.reg), (Rin.nR - n_shift)) != Q_PASS) {
            whereami(__FILE__, __LINE__);
            return Q_FAIL;
          }
        } else {
          if (fill_device_memory<T>((T *)(Rout.reg) + (Rin.nR + n_shift), size_t(-n_shift), value) != Q_PASS) {
            whereami(__FILE__, __LINE__);
            return Q_FAIL;
          }
          if (copy_device_memory<T>((T *)(Rout.reg), (T const *)(Rin.reg) - n_shift, (Rin.nR + n_shift)) != Q_PASS) {
            whereami(__FILE__, __LINE__);
            return Q_FAIL;
          }
        }
      }
      return Q_PASS;
    }
  };

  class f1_shift_f2 {
  public:
    f1_shift_f2() {
      addop(I1, new __f1_shift_f2<Q_I1_t>);
      addop(I2, new __f1_shift_f2<Q_I2_t>);
      addop(I4, new __f1_shift_f2<Q_I4_t>);
      addop(I8, new __f1_shift_f2<Q_I8_t>);
    }

    int operator()(gpu_register const & Rin,
                   gpu_register & Rout,
                   int n_shift,
                   Q_I8_t fill_value) {
      if (op.find(Rin.fldtype) == op.end()) {
        cerr << "f1_shift_f2 not implemented for "
             << get_fldtype_string(Rin.fldtype) << endl;
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      return op[Rin.fldtype]->operator()(Rin, Rout, n_shift, fill_value);
    }

  private:
    template <typename T>
    inline
    void addop(FLD_TYPE index, T * oper) {
      op[index] = shared_ptr<__f1_shift_f2_interface>
        (static_cast<__f1_shift_f2_interface *>(oper));
    }
    map<FLD_TYPE, shared_ptr<__f1_shift_f2_interface> > op;
  };

  static f1_shift_f2 s_f1_shift_f2;

  int f1_shift_f2_GPU(gpu_register const & Rin,
                      gpu_register & Rout,
                      int n_shift,
                      Q_I8_t fill_value) {
    return s_f1_shift_f2(Rin, Rout, n_shift, fill_value);
  }

}

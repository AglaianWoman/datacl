#include "hip/hip_runtime.h"
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <unistd.h>

#include <iostream>
#include <iterator>
#include <string>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <map>
#include <functional>
#include <boost/shared_ptr.hpp>
#include <boost/lexical_cast.hpp>

#include "q_gpu_server.hpp"
#include "q_gpu.hpp"
#include "q_gpu_server_kernels.hpp"

#include "q_utilities.hpp"
#include "q_gpu_registers.hpp"
#include "q_gpu_f1s1opf2.hpp"
#include "q_gpu_f1f2opf3.hpp"
#include "q_gpu_f_to_s.hpp"
#include "q_gpu_count.hpp"
#include "q_gpu_f1_shift_f2.hpp"
#include "q_gpu_srt_join.hpp"
#include "q_gpu_sort.hpp"
#include "q_gpu_index.hpp"
#include "q_gpu_funnel.hpp"

using namespace std;


namespace QDB {

  //==================================================================
  //Operations on GPU
  //==================================================================

  int
  f1s1opf2_GPU(
               string const & d_fld1,
               string const & str_scalar,
               string const & str_op,
               string const & d_fld2
               ) {
    if (!GPU::getInstance().is_valid_register(d_fld1) ||
        !GPU::getInstance().is_valid_register(d_fld2)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return f1s1opf2_GPU(GPU::getInstance()[d_fld1], str_scalar, str_op, GPU::getInstance()[d_fld2]);
  }

  int
  f1f2opf3_GPU(
               string const & d_fld1,
               string const & d_fld2,
               string const & str_op,
               string const & d_fld3
               ) {
    if (!GPU::getInstance().is_valid_register(d_fld1) ||
        !GPU::getInstance().is_valid_register(d_fld2) ||
        !GPU::getInstance().is_valid_register(d_fld3)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return f1f2opf3_GPU(GPU::getInstance()[d_fld1], GPU::getInstance()[d_fld2], str_op, GPU::getInstance()[d_fld3]);
  }

  int
  f_to_s_GPU(
             string const & d_fld1,
             string const & str_op,
             string & str_result
             ) {
    if (!GPU::getInstance().is_valid_register(d_fld1)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return f_to_s_GPU(GPU::getInstance()[d_fld1], NULL, str_op, str_result);
  }

  int
  f1f2_to_s_GPU(
                string const & d_fld1,
                string const & d_fld2,
                string const & str_op,
                string & str_result
                ) {
    if (!GPU::getInstance().is_valid_register(d_fld1) ||
        !GPU::getInstance().is_valid_register(d_fld2)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }

    // note: f1f2_to_s_GPU is same as f_to_s_GPU with nn field for f
    // f1 (Q_I1_t) <--> nn (Q_I1_t), f2 <--> f
    // f1f2_to_s_GPU assumes there is NO nn field for f
    return f_to_s_GPU(GPU::getInstance()[d_fld2], &GPU::getInstance()[d_fld1], str_op, str_result);
  }

  //shift f1 by shift_amt (can be +ve or -ve) and get f2. fill in given value at shifted spots
  int
  f1_shift_f2_GPU(
                  string const & d_fld1,
                  string const & d_fld2,
                  string const & str_shift_amt,
                  string const & str_newval
                  /* new value inserted in null spots created by shift */
                  ) {
    int shift_amt;
    try {
      shift_amt = boost::lexical_cast<int>(str_shift_amt);
    } catch (boost::bad_lexical_cast const &) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }

    Q_I8_t fill_value;
    try {
      fill_value = boost::lexical_cast<Q_I8_t>(str_newval);
    } catch (boost::bad_lexical_cast const &) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }

    if (!GPU::getInstance().is_valid_register(d_fld1) ||
        !GPU::getInstance().is_valid_register(d_fld2)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return f1_shift_f2_GPU(GPU::getInstance()[d_fld1],
                           GPU::getInstance()[d_fld2],
                           shift_amt,
                           fill_value);
  }

  int
  count_GPU(
            string const & d_idx,
            string const & d_cnd,
            string const & d_dst,
            string const & safe_mode
            ) {
    if (!GPU::getInstance().is_valid_register(d_idx) ||
        (!d_cnd.empty() && !GPU::getInstance().is_valid_register(d_cnd)) ||
        !GPU::getInstance().is_valid_register(d_dst)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return count_GPU(GPU::getInstance()[d_idx],
                     (d_cnd.empty() ? NULL : &GPU::getInstance()[d_cnd]),
                     GPU::getInstance()[d_dst],
                     (safe_mode.empty() || (safe_mode != string("false"))));
  }

  int
  countf_GPU(
             string const & d_idx,
             string const & d_cnd,
             string const & d_val,
             string const & d_dst,
             string const & safe_mode
             ) {
    if (!GPU::getInstance().is_valid_register(d_idx) ||
        (!d_cnd.empty() && !GPU::getInstance().is_valid_register(d_cnd)) ||
        !GPU::getInstance().is_valid_register(d_val) ||
        !GPU::getInstance().is_valid_register(d_dst)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return countf_GPU(GPU::getInstance()[d_idx],
                      (d_cnd.empty() ? NULL : &GPU::getInstance()[d_cnd]),
                      GPU::getInstance()[d_val],
                      GPU::getInstance()[d_dst],
                      (safe_mode.empty() || (safe_mode != string("false"))));
  }

  //join - for all elements in vs that are in ls,
  // copy the corresponding value of vs to vd and mark nnd to 1.
  // for elements that are in vs and are not found in ls, mark vd to 0 and also nnd to 0.
  int
  join_GPU(
           string const & d_ls, // ls = link source
           string const & d_vs, // vs = value source
           string const & d_ld, // ld = link destination
           string const & d_vd, // vd = value destination
           string const & d_vd_nn, // vd_nn = nn field for value destination
           string const & str_op // <reg | min | max | sum | cnt | and | or>
           ) {
    if (!GPU::getInstance().is_valid_register(d_ls) ||
        !GPU::getInstance().is_valid_register(d_vs) ||
        !GPU::getInstance().is_valid_register(d_ld) ||
        !GPU::getInstance().is_valid_register(d_vd) ||
        !GPU::getInstance().is_valid_register(d_vd_nn)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return srt_join_GPU(GPU::getInstance()[d_ls],
                        GPU::getInstance()[d_vs],
                        GPU::getInstance()[d_ld],
                        str_op, GPU::getInstance()[d_vd],
                        GPU::getInstance()[d_vd_nn]);
  }

  // sorts in-place and will change the contents of the specified register
  int
  sort_GPU(
           string const & d_fld,
           string const & order
           ) {
    if (!GPU::getInstance().is_valid_register(d_fld)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return sort_GPU(GPU::getInstance()[d_fld], order);
  }

  int
  sortf1f2_GPU(
               string const & d_key,
               string const & d_val,
               string const & srttype
               ) {
    if (!GPU::getInstance().is_valid_register(d_key) ||
        !GPU::getInstance().is_valid_register(d_val)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return sortf1f2_GPU(GPU::getInstance()[d_key], GPU::getInstance()[d_val], srttype);
  }

  int
  mk_idx_GPU(
             string const & d_idx
             ) {
    if (!GPU::getInstance().is_valid_register(d_idx)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return mk_idx_GPU(GPU::getInstance()[d_idx]);
  }

  int
  permute_by_idx_GPU(
                     string const & d_data,
                     string const & d_idx,
                     string const & d_direction,
                     string const & d_result
                     ) {
    if (!GPU::getInstance().is_valid_register(d_data) ||
        !GPU::getInstance().is_valid_register(d_idx) ||
        !GPU::getInstance().is_valid_register(d_result)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return permute_by_idx_GPU(GPU::getInstance()[d_data],
                              GPU::getInstance()[d_idx],
                              (d_direction != string("reverse")),
                              GPU::getInstance()[d_result]);
  }

  int
  funnel_GPU(
             string const & d_key,
             string const & str_value,
             string const & d_same_session,
             string const & d_prev_funnel,
             string const & d_result
             ) {
    if (!GPU::getInstance().is_valid_register(d_key) ||
        !GPU::getInstance().is_valid_register(d_same_session) ||
        !GPU::getInstance().is_valid_register(d_prev_funnel) ||
        !GPU::getInstance().is_valid_register(d_result)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return funnel_GPU(GPU::getInstance()[d_key],
                      str_value,
                      GPU::getInstance()[d_same_session],
                      GPU::getInstance()[d_prev_funnel],
                      GPU::getInstance()[d_result]);
  }

  int
  funnel_count_GPU(
                   string const & d_key,
                   string const & str_distance,
                   string const & d_same_session,
                   string const & d_curr_funnel,
                   string const & d_result
                   ) {
    if (!GPU::getInstance().is_valid_register(d_key) ||
        !GPU::getInstance().is_valid_register(d_same_session) ||
        !GPU::getInstance().is_valid_register(d_curr_funnel) ||
        !GPU::getInstance().is_valid_register(d_result)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    size_t distance;
    try {
      distance = boost::lexical_cast<size_t>(str_distance);
    } catch (boost::bad_lexical_cast const &) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return funnel_count_GPU(GPU::getInstance()[d_key],
                            distance,
                            GPU::getInstance()[d_same_session],
                            GPU::getInstance()[d_curr_funnel],
                            GPU::getInstance()[d_result]);
  }

}

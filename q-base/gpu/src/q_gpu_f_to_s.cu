#include "hip/hip_runtime.h"
#include <limits>
#include <iostream>
#include <sstream>
#include <string>
#include <map>
#include <boost/shared_ptr.hpp>
#include "q_types.hpp"
#include "q_gpu_registers.hpp"
#include "q_utilities.hpp"
#include "q_gpu_server_kernels.hpp"
#include "q_gpu_f_to_s.hpp"

using std::string;
using std::map;
using std::cerr;
using std::endl;
using std::numeric_limits;
using std::ostringstream;

using boost::shared_ptr;

#include <sys/time.h>

namespace QDB {

  class GPU_timer {
  public:
    GPU_timer(string const & nm)
      : name(nm),
        start(),
        end() {
      gettimeofday(&start, NULL);
    }
    ~GPU_timer() {
      gettimeofday(&end, NULL);
      double diff = ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec));
      std::cout << "Time Taken for " << name << ": " << diff << " usec" << std::endl;
    }

  private:
    string name;
    struct timeval start, end;
  };

  struct __f_to_s_interface {
    __f_to_s_interface() {
    }

    virtual ~__f_to_s_interface() {
    }

    virtual int operator()(gpu_register const & R,
                           gpu_register const * pRfilter1,
                           gpu_register const * pRfilter2,
                           string & str_result) {
      cerr << "ERROR: NOT IMPLEMENTED" << endl;
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
  };

  template <typename T,
            typename T_RESULT,
            typename OP_TYPE>
  struct __f_to_s : public __f_to_s_interface {
    __f_to_s(T_RESULT init_val = 0)
      : __f_to_s_interface(),
        m_initVal(init_val) {
    }

    virtual ~__f_to_s() {
    }

    virtual int operator()(gpu_register const & R,
                           gpu_register const * pRfilter1,
                           gpu_register const * pRfilter2,
                           string & str_result) {
      T_RESULT * pPartial;
      if (hipMalloc((void **)(&pPartial), NBLOCKS * sizeof(T_RESULT)) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      {
        dim3 dimGrid(NBLOCKS);
        dim3 dimBlock(NTHREADS);
        size_t sharedMemSize = dimBlock.x * sizeof(T_RESULT);
        if (sharedMemSize > MAX_SHARED_MEM) {
          whereami(__FILE__, __LINE__);
          return Q_FAIL;
        }
        filter_reduce_to_partial_kernel<T, T_RESULT, OP_TYPE>
          <<< dimGrid, dimBlock, sharedMemSize >>>
          (static_cast<T const *>(R.reg),
           size_t(R.nR),
           pPartial,
           static_cast<Q_I1_t const *>(pRfilter1 ? pRfilter1->reg : NULL),
           static_cast<Q_I1_t const *>(pRfilter2 ? pRfilter2->reg : NULL),
           OP_TYPE(),
           m_initVal);
      }
      {
        dim3 dimGrid(1);
        dim3 dimBlock(NTHREADS);
        reduce_partial_results_kernel<T_RESULT, OP_TYPE>
          <<< dimGrid, dimBlock >>>
          (pPartial,
           NBLOCKS,
           OP_TYPE(),
           m_initVal);
        T_RESULT result;
        if (hipMemcpy((void *)(&result), (void const *)(pPartial), sizeof(T_RESULT), hipMemcpyDeviceToHost) != hipSuccess) {
          whereami(__FILE__, __LINE__);
          return Q_FAIL;
        }
        str_result = static_cast<ostringstream*>( &(ostringstream() << result) )->str();
      }
      if (hipFree(pPartial) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      return Q_PASS;
    }
    T_RESULT const m_initVal;
  };

  // specialization for summation

  template <typename T>
  struct __f_to_s<T, Q_I8_t, ADD<Q_I8_t> > : public __f_to_s_interface {
    __f_to_s()
      : __f_to_s_interface() {
    }

    virtual ~__f_to_s() {
    }

    virtual int operator()(gpu_register const & R,
                           gpu_register const * pRfilter1,
                           gpu_register const * pRfilter2,
                           string & str_result) {
      unsigned long long int * d_sum;
      if (hipMalloc((void **)(&d_sum), sizeof(unsigned long long int)) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      if (hipMemset(d_sum, 0, sizeof(unsigned long long int)) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }

      dim3 dimGrid(NBLOCKS);
      dim3 dimBlock(NTHREADS);
      size_t sharedMemSize = dimBlock.x * sizeof(Q_I8_t);
      if (sharedMemSize > MAX_SHARED_MEM) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      filter_sum_kernel<T><<< dimGrid, dimBlock, sharedMemSize >>>
        ((T *)(R.reg), size_t(R.nR),
         (Q_I1_t const *)(pRfilter1 ? pRfilter1->reg : NULL),
         (Q_I1_t const *)(pRfilter2 ? pRfilter2->reg : NULL),
         d_sum);

      unsigned long long int tmp_result;
      if (hipMemcpy((void *)(&tmp_result), (void const *)(d_sum), sizeof(unsigned long long int), hipMemcpyDeviceToHost) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }

      Q_I8_t result = Q_I8_t(tmp_result);
      str_result = static_cast<ostringstream*>( &(ostringstream() << result) )->str();
      if (hipFree(d_sum) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      return Q_PASS;
    }
  };

  struct f_to_s {
  public:
    f_to_s() {
      add_integer_ops<I1>();
      add_integer_ops<I2>();
      add_integer_ops<I4>();
      add_integer_ops<I8>();
    }

    int operator()(gpu_register const & R,
                   gpu_register const * pRnn,
                   string const & str_op,
                   string & result) {
      if ((op.find(str_op) == op.end()) ||
          (op[str_op].find(R.fldtype) == op[str_op].end())) {
        cerr << "ERROR: f_to_s and f1f2_to_s not implemented for "
             << "type(f)=" << get_fldtype_string(R.fldtype) << " "
             << "op=" << str_op << endl;
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      return op[str_op][R.fldtype]->operator()(R, pRnn, NULL, result);
    }

  private:

    template <typename T>
    inline
    void addop(string const & opname, FLD_TYPE ftype, T * oper) {
      op[opname][ftype] = shared_ptr<__f_to_s_interface>(static_cast<__f_to_s_interface *>(oper));
    }

    template <FLD_TYPE ftype>
    inline
    void add_integer_ops() {
      typedef typename fldtype_traits<ftype>::q_type q_type;
      addop("min", ftype, new __f_to_s<q_type, q_type, MIN<q_type> >(numeric_limits<q_type>::max()));
      addop("max", ftype, new __f_to_s<q_type, q_type, MAX<q_type> >(numeric_limits<q_type>::min()));
      addop("sum", ftype, new __f_to_s<q_type, Q_I8_t, ADD<Q_I8_t> >());
    }

    typedef map<FLD_TYPE, shared_ptr<__f_to_s_interface> > map_ftype_to_oper;
    typedef map<string, map_ftype_to_oper> map_opname_to_ftype;
    map_opname_to_ftype op;
  };

  static f_to_s s_f_to_s;

  int f_to_s_GPU(gpu_register const & R,
                      gpu_register const * pRnn,
                      string const & str_op,
                      string & result) {
    return s_f_to_s(R, pRnn, str_op, result);
  }
}

#include <string>
#include <iostream>
#include <map>
#include <functional>
#include <boost/shared_ptr.hpp>
#include <boost/lexical_cast.hpp>
#include "q_types.hpp"
#include "q_gpu_registers.hpp"
#include "q_utilities.hpp"
#include "q_gpu_funnel.hpp"
#include "q_gpu_server_kernels.hpp"

using std::string;
using std::cerr;
using std::endl;
using std::map;

using std::ptr_fun;
using boost::shared_ptr;
using boost::lexical_cast;
using boost::bad_lexical_cast;

namespace QDB {

  struct __funnel_interface {
    __funnel_interface () {
    }

    virtual ~__funnel_interface () {
    }

    virtual int operator()(gpu_register const & Rkey,
                           string const & str_value,
                           gpu_register const & Rsame_session,
                           gpu_register const & Rprev_funnel,
                           gpu_register const & Rresult) {
      cerr << "ERROR: NOT IMPLEMENTED" << endl;
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
  };

  template <typename T>
  struct __funnel : public __funnel_interface {
    __funnel()
      : __funnel_interface() {
    }

    virtual ~__funnel() {
    }

    virtual int operator()(gpu_register const & Rkey,
                           string const & str_value,
                           gpu_register const & Rsame_session,
                           gpu_register const & Rprev_funnel,
                           gpu_register const & Rresult) {
      dim3 dimGrid(NBLOCKS);
      dim3 dimBlock(NTHREADS);
      T value;
      try {
        value = lexical_cast<T>(str_value);
      } catch (bad_lexical_cast const &) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      funnel_kernel<T>
        <<< dimGrid, dimBlock >>>
        ((T const *)(Rkey.reg), size_t(Rkey.nR),
         value,
         (Q_I1_t const *)(Rsame_session.reg),
         (Q_I1_t const *)(Rprev_funnel.reg),
         (Q_I1_t *)(Rresult.reg));
      return Q_PASS;
    }
  };

  struct funnel {
  public:
    funnel() {
      addop<I1>();
      addop<I2>();
      addop<I4>();
      addop<I8>();
    }

    virtual int operator()(gpu_register const & Rkey,
                           string const & str_value,
                           gpu_register const & Rsame_session,
                           gpu_register const & Rprev_funnel,
                           gpu_register const & Rresult) {
      if (op.find(Rkey.fldtype) == op.end()) {
        cerr << "ERROR: funnel not supported for "
             << "type(idx)=" << get_fldtype_string(Rkey.fldtype) << endl;
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      return op[Rkey.fldtype]->operator()(Rkey, str_value, Rsame_session, Rprev_funnel, Rresult);
    }

  private:
    template <FLD_TYPE index>
    inline
    void addop() {
      typedef typename fldtype_traits<index>::q_type q_type;
      op[index] = shared_ptr<__funnel_interface>
        (static_cast<__funnel_interface *>(new __funnel<q_type>));
    }
    map<FLD_TYPE, shared_ptr<__funnel_interface> > op;
  };

  static funnel s_funnel;

  int
  funnel_GPU(gpu_register const & Rkey,
             string const & str_value,
             gpu_register const & Rsame_session,
             gpu_register const & Rprev_funnel,
             gpu_register & Rresult) {
    return s_funnel(Rkey, str_value, Rsame_session, Rprev_funnel, Rresult);
  }

  struct __funnel_count_interface {
    __funnel_count_interface () {
    }

    virtual ~__funnel_count_interface () {
    }

    virtual int operator()(gpu_register const & Rkey,
                           size_t distance,
                           gpu_register const & Rsame_session,
                           gpu_register const & Rprev_funnel_count,
                           gpu_register & Rresult) {
      cerr << "ERROR: NOT IMPLEMENTED" << endl;
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
  };

  template <typename T_INDEX, typename T_RESULT>
  struct __funnel_count : public __funnel_count_interface {
    __funnel_count()
      : __funnel_count_interface() {
    }

    virtual ~__funnel_count() {
    }

    virtual int operator()(gpu_register const & Rkey,
                           size_t distance,
                           gpu_register const & Rsame_session,
                           gpu_register const & Rcurr_funnel,
                           gpu_register & Rresult) {
      if (hipMemset(Rresult.reg, 0, Rresult.nR * sizeof(T_RESULT)) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }

      dim3 dimGrid(NBLOCKS);
      dim3 dimBlock(NTHREADS);

      size_t sharedMemSize = Rresult.nR * sizeof(T_RESULT);
      if (sharedMemSize > MAX_SHARED_MEM) {
        sharedMemSize = 0;
      }

      funnel_count_kernel<T_INDEX, T_RESULT>
        <<< dimGrid, dimBlock, sharedMemSize >>>
        ((T_INDEX const *)(Rkey.reg),
         (Q_I1_t const *)(Rsame_session.reg),
         (Q_I1_t const *)(Rcurr_funnel.reg),
         distance,
         (T_RESULT *)(Rresult.reg),
         size_t(Rkey.nR),
         size_t(Rresult.nR),
         sharedMemSize);
      return Q_PASS;
    }
  };

  struct funnel_count {
  public:
    funnel_count() {
      addops<I1>();
      addops<I2>();
      addops<I4>();
      addops<I8>();
    }

    int operator()(gpu_register const & Rkey,
                   size_t distance,
                   gpu_register const & Rsame_session,
                   gpu_register const & Rprev_funnel_count,
                   gpu_register & Rresult) {
      if ((op.find(Rkey.fldtype) == op.end()) ||
          (op[Rkey.fldtype].find(Rresult.fldtype) == op[Rkey.fldtype].end())) {
        cerr << "ERROR: funnel_count not supported for"
             << " type(key)=" << get_fldtype_string(Rkey.fldtype)
             << " type(result)=" << get_fldtype_string(Rresult.fldtype)
             << endl;
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      return op[Rkey.fldtype][Rresult.fldtype]->operator()(Rkey, distance, Rsame_session, Rprev_funnel_count, Rresult);
    }

  private:
    typedef map<FLD_TYPE, shared_ptr<__funnel_count_interface> > map_resultfld_to_oper;
    typedef map<FLD_TYPE, map_resultfld_to_oper> map_keyfld_to_resultfld;

    template <FLD_TYPE key, FLD_TYPE result>
    inline
    void addop() {
      typedef typename fldtype_traits<key>::q_type key_type;
      typedef typename fldtype_traits<result>::q_type result_type;
      op[key][result] = shared_ptr<__funnel_count_interface>
        (static_cast<__funnel_count_interface *>(new __funnel_count<key_type, result_type>));
    }

    template <FLD_TYPE key>
    inline
    void addops() {
      //      addop<key, I1>();
      //      addop<key, I2>();
      addop<key, I4>();
      addop<key, I8>();
    }

    map_keyfld_to_resultfld op;
  };

  static funnel_count s_funnel_count;

  int
  funnel_count_GPU(gpu_register const & Rkey,
                   size_t distance,
                   gpu_register const & Rsame_session,
                   gpu_register const & Rcurr_funnel,
                   gpu_register & Rresult) {
    return s_funnel_count(Rkey, distance, Rsame_session, Rcurr_funnel, Rresult);
  }
};

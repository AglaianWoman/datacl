#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <sstream>
#include <fstream>
#include <map>
#include <boost/shared_ptr.hpp>
#include <boost/lexical_cast.hpp>
#include <boost/filesystem.hpp>

#include "q_gpu_registers.hpp"
#include "q_types.hpp"
#include "q_utilities.hpp"

#define DEFAULT_GPU_PORT 8080

using std::string;
// using std::stringstream;
using std::ostringstream;
using std::ostream;
using std::ofstream;
using std::endl;
using std::map;
using std::cerr;
using std::endl;

using boost::shared_ptr;

static inline string safe_getenv(char const * name) {
  char * val = getenv(name);
  return (val ? string(val) : string(""));
}

#ifdef SUPPORT_LEGACY
namespace QDB_LEGACY {
  typedef enum _FLD_TYPE {
    undef_fldtype,
    B,
    I1,
    I2,
    I4,
    I8,
    F4,
    F8,
    SC, /* raw string. Must be fixed length. len must be set */
    SV, /* Variable length string. dict_tbl_id must be set. */
  } FLD_TYPE;

  const size_t MAX_LEN_FILE_NAME = 511;
  const size_t MAX_LEN_DIR_NAME = 511;
  const size_t MAX_LEN_FLD_NAME = 31;
  const size_t MAX_LEN_TBL_NAME = 31;

  typedef struct _GPU_REG_TYPE {
    void * reg; // points to data on device
    size_t filesz;
    bool is_busy;
    long long nR;
    FLD_TYPE fldtype;
    char filename[MAX_LEN_DIR_NAME+63];
    char tbl[MAX_LEN_TBL_NAME+1];
    char h_fld[MAX_LEN_FLD_NAME+1];
    char d_fld[MAX_LEN_FLD_NAME+1];
  } GPU_REG_TYPE;
}
#endif // SUPPORT_LEGACY

namespace QDB {
  // ************
  // gpu_register
  // ************

  gpu_register::gpu_register()
    : filename(),
      tbl(),
      h_fld(),
      d_fld(),
      nR(0),
      fldtype(undef_fldtype),
      pReg(),
      reg(NULL) {
  }

  int
  gpu_register::init(size_t in_nR, FLD_TYPE in_fldtype, string const & in_filename,
                     string const & in_tbl, string const & in_h_fld, string const & in_d_fld) {
    filename = in_filename;
    tbl = in_tbl;
    h_fld = in_h_fld;
    d_fld = in_d_fld;
    if (!pReg.get() || (nR != in_nR) || (fldtype != in_fldtype)) {
      pReg.reset((void *)(NULL));
      reg = NULL;
      nR = in_nR;
      fldtype = in_fldtype;
      size_t filesz = nR * get_sizeof(fldtype);
      void * mem;
      if (hipMalloc(&mem, filesz) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      pReg = boost::shared_ptr<void>(mem, std::ptr_fun(hipFree));
      reg = pReg.get();
    }
    return Q_PASS;
  }

  // list register properties/metadata
  int
  gpu_register::describe(
                         string & rsltbuf
                         ) const {
    ostringstream ss;
    ss << get_fldtype_string(fldtype)
       << "," << nR
       << "," << tbl
       << "," << h_fld
       << "," << d_fld;
    rsltbuf += ss.str();
    return Q_PASS;
  }

  int
  gpu_register::load(string const & in_filename) {
    if (!in_filename.empty()) {
      filename = in_filename;
    }

    if (filename.empty() || !boost::filesystem::exists(filename)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;      
    }

    MmappedData Data(filename);
    if (!Data.is_ok()) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }

    if (!pReg.get() || (nR != (Data.nX() / get_sizeof(fldtype)))) {
      pReg.reset((void *)(NULL));
      reg = NULL;
      nR = Data.nX() / get_sizeof(fldtype);
    }

    if (nR) {
      if (!reg) {
        void * mem;
        if (hipMalloc(&mem, Data.nX()) != hipSuccess) {
          whereami(__FILE__, __LINE__);
          return Q_FAIL;
        }
        pReg = boost::shared_ptr<void>(mem, std::ptr_fun(hipFree));
        reg = pReg.get();
      }
      if (hipMemcpy(reg, Data.X(), Data.nX(), hipMemcpyHostToDevice) != hipSuccess) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
    }

    return Q_PASS;
  }

  int
  gpu_register::store(string const & in_filename) const {
    if (!in_filename.empty()) {
      filename = in_filename;
    }
    MmappedData Data(filename, nR * get_sizeof(fldtype));
    if (!Data.is_ok()) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    if (hipMemcpy(Data.X(), reg, Data.nX(), hipMemcpyDeviceToHost) != hipSuccess) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return Q_PASS;
  }

  // ************
  // print_helper
  // ************

  struct __print_helper_interface {
    virtual int operator()(ostream & os, string const & separator,
                           gpu_register const & reg, size_t nelem, bool bTail) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
  };

  template <typename T>
  struct __print_helper : public __print_helper_interface {
    virtual int operator()(ostream & os, string const & separator,
                           gpu_register const & reg, size_t nelem, bool bTail) {
      if (reg.nR == 0) {
        return Q_FAIL;
      }
      nelem = nelem ? std::min(size_t(reg.nR), nelem) : size_t(reg.nR);
      size_t numbytes = nelem * sizeof(T);
      void * dst = malloc(numbytes);
      T const * src = (T *)(reg.reg);
      if (bTail) {
        src += (reg.nR - nelem);
      }
      hipMemcpy(dst, (void *)(src), numbytes, hipMemcpyDeviceToHost);
      T * i = (T *)(dst);
      T * iEnd = i + nelem;
      typedef typename qtype_traits<T>::print_type print_type;
      os << static_cast<print_type>(*i);
      while (++i != iEnd) {
        os << separator << static_cast<print_type>(*i);
      }
      free(dst);
      return Q_PASS;
    }
  };


  struct print_helper {
    int operator()(ostream & os, string const & separator,
                   gpu_register const & reg, size_t nelem = 0, bool bTail = false) {
      if (op.find(reg.fldtype) == op.end()) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      op[reg.fldtype]->operator()(os, separator, reg, nelem, bTail);
      return Q_PASS;
    }

    print_helper() {
      addop<I1>();
      addop<I2>();
      addop<I4>();
      addop<I8>();
      addop<F4>();
      addop<F8>();
    }

    template<FLD_TYPE fldtype>
    inline void addop() {
      typedef typename fldtype_traits<fldtype>::q_type q_type;
      op[fldtype] = shared_ptr<__print_helper_interface>
        (static_cast<__print_helper_interface *>(new __print_helper<q_type>()));
    }

    map<FLD_TYPE, shared_ptr<__print_helper_interface> > op;
  };

  static print_helper s_print_helper;

  // ***
  // GPU
  // ***

  //get the free memory in bytes available on the GPU
  int
  GPU::list_free_mem(string & str_free_mem) const {
    size_t available, total;
    if (hipMemGetInfo(&available, &total) != hipSuccess) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    str_free_mem += boost::lexical_cast<string>(available);
    return Q_PASS;
  }

  bool GPU::is_valid_register(string const & reg_name) const {
    return (gpu_reg.find(reg_name) != gpu_reg.end());
  }

  // returns the number of registers
  int
  GPU::list_num_registers(string & str_num_reg) const {
    str_num_reg = boost::lexical_cast<string>(gpu_reg.size());
    return Q_PASS;
  }

  void
  GPU::list_registers(string & str_free_reg) const {
    if (!gpu_reg.empty()) {
      ostringstream ss;
      map<string, gpu_register>::const_iterator i = gpu_reg.begin();
      ss << i->first;
      for (++i; i != gpu_reg.end(); ++i) {
        ss << ":" << i->first;
      }
      str_free_reg += ss.str();
    }
  }

  // list register properties/metadata
  int
  GPU::describe_register(string const & str_reg, string & rsltbuf) const {
    map<string, gpu_register>::const_iterator iFound = gpu_reg.find(str_reg);
    if (iFound == gpu_reg.end()) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    return iFound->second.describe(rsltbuf);
  }

  //Prints the (first or last) N data elements of the specified register
  int
  GPU::print_reg_data (string const & str_reg, string const & str_nelem,
                       string const & str_head_or_tail, string & rsltbuf) {
    if (str_head_or_tail != "head" &&
        str_head_or_tail != "tail") {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }

    map<string, gpu_register>::const_iterator iFound = gpu_reg.find(str_reg);
    if (iFound == gpu_reg.end()) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }

    int nelem = atoi(str_nelem.c_str());
    if (( nelem <= 0 ) || ( nelem > iFound->second.nR )) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }

    ostringstream sout;
    if (s_print_helper(sout, "\t", iFound->second, nelem, (str_head_or_tail == "tail")) != Q_PASS) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    rsltbuf += sout.str();

    return Q_PASS;
  }

  //prints data in given register to the given ascii file. Only used for debugging. currently assuming int datatype only.
  int
  GPU::print_data_to_ascii_file (string const & str_reg,
                                 string const & filename) {
    ofstream os(filename.c_str());
    if (s_print_helper(os, "\n", gpu_reg[str_reg], 0, false) != Q_PASS) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    os << endl;
    return Q_PASS;
  }

  //adds a new field from the given file, and also sets appropriate metadata
  int
  GPU::add_fld(
               string const & str_filesz,
               string const & str_nR,
               string const & str_fldtype,
               string const & filename,
               string const & tbl,
               string const & h_fld,
               string const & d_fld
               ) {
    FLD_TYPE fldtype = get_fldtype_enum(str_fldtype);
    if ((fldtype != I1) && (fldtype != I2) && (fldtype != I4) && (fldtype != I8) &&
        (fldtype != F4) && (fldtype != F8)) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }

    size_t nR;
    if (!str_nR.empty()) {
      try {
        nR = boost::lexical_cast<size_t>(str_nR);
      } catch (boost::bad_lexical_cast const &) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
    } else if (!str_filesz.empty()) {
      try {
        size_t filesz = boost::lexical_cast<size_t>(str_nR);
        nR = filesz / get_sizeof(fldtype);
      } catch (boost::bad_lexical_cast const &) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
    } else if (!filename.empty() && boost::filesystem::exists(filename) && boost::filesystem::file_size(filename)) {
      size_t filesz = boost::filesystem::file_size(filename);
      nR = filesz / get_sizeof(fldtype);
    } else {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }

    if (gpu_reg[d_fld].init(nR, fldtype, filename, tbl, h_fld, d_fld) != Q_PASS) {
      del_fld(d_fld);
      whereami(__FILE__, __LINE__);
      return Q_FAIL;      
    }

#ifdef SUPPORT_LEGACY
    create_legacy_mmap_file();
#endif // SUPPORT_LEGACY

    return Q_PASS;
  }

  //deletes given field
  void
  GPU::del_fld(
               string const & d_fld
               ) {
    map<string, gpu_register>::iterator iFound = gpu_reg.find(d_fld);
    if (iFound != gpu_reg.end()) {
      gpu_reg.erase(iFound);
#ifdef SUPPORT_LEGACY
      create_legacy_mmap_file();
#endif // SUPPORT_LEGACY
    } else if (d_fld == string("ALL_FLDS")) {
      gpu_reg.clear();
#ifdef SUPPORT_LEGACY
      create_legacy_mmap_file();
#endif // SUPPORT_LEGACY
    }
  }

  // transfers data from file to GPU register, and sets metadata
  int
  GPU::load(
            string const & str_filesz,
            string const & str_nR,
            string const & str_fldtype,
            string const & filename,
            string const & tbl,
            string const & h_fld,
            string const & d_fld
            ) {
    if ((add_fld(str_filesz, str_nR, str_fldtype, filename, tbl, h_fld, d_fld) != Q_PASS) ||
        (gpu_reg[d_fld].load() != Q_PASS)) {
      del_fld(d_fld);
      return Q_FAIL;
    }
    return Q_PASS;
  }

  // transfers data from register on device to file
  int
  GPU::store(string const & filename,
             string const & d_fld) {
    map<string, gpu_register>::const_iterator iFound = gpu_reg.find(d_fld);
    if (iFound == gpu_reg.end()) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    iFound->second.store(filename);
#ifdef SUPPORT_LEGACY
    create_legacy_mmap_file();
#endif // SUPPORT_LEGACY
    return Q_PASS;
  }

  int
  GPU::swap_flds(string const & d_fld1,
                 string const & d_fld2) {
    bool bValid1 = is_valid_register(d_fld1);
    bool bValid2 = is_valid_register(d_fld2);
    if (bValid1 && bValid2) {
      std::swap(gpu_reg[d_fld1], gpu_reg[d_fld2]);
    } else if (bValid1) {
      gpu_reg[d_fld2] = gpu_reg[d_fld1];
      del_fld(d_fld1);
    } else if (bValid2) {
      gpu_reg[d_fld1] = gpu_reg[d_fld2];
      del_fld(d_fld2);
    } else {
      return Q_FAIL;
    }
    return Q_PASS;
  }

  gpu_register &
  GPU::operator[](string const & str_reg) {
    return gpu_reg[str_reg];
  }

  GPU & GPU::getInstance() {
    static GPU instance;
    return instance;
  }

  GPU::GPU()
    : gpu_server(safe_getenv("Q_GPU_SERVER")),
      docroot(safe_getenv("Q_DOCROOT")),
      data_dir(safe_getenv("Q_DATA_DIR")),
      cwd(),
      gpu_port(DEFAULT_GPU_PORT),
      ddir_id(-1),
      alt_ddir_id(-1),
      gpu_reg() {
    string port = safe_getenv("Q_GPU_PORT");
    if (!port.empty()) {
      gpu_port = boost::lexical_cast<int>(port);
    }
    char * cwd_tmp = getcwd(NULL, 0);
    if (cwd_tmp) {
      cwd = string(cwd_tmp);
      free(cwd_tmp);
    }
  }

#ifdef SUPPORT_LEGACY
  int
  GPU::create_legacy_mmap_file() const {
    string gpu_meta_file = docroot + string("/docroot.gpu");
    MmappedData Data(gpu_meta_file, sizeof(QDB_LEGACY::GPU_REG_TYPE) * gpu_reg.size());
    if (!Data.is_ok()) {
      whereami(__FILE__, __LINE__);
      return Q_FAIL;
    }
    QDB_LEGACY::GPU_REG_TYPE * pRegOld = (QDB_LEGACY::GPU_REG_TYPE *)(Data.X());
    for (map<string, gpu_register>::const_iterator i = gpu_reg.begin(), iEnd = gpu_reg.end(); i != iEnd; ++i, ++pRegOld) {
      pRegOld->reg = NULL;
      pRegOld->filesz = i->second.nR * get_sizeof(i->second.fldtype);
      pRegOld->is_busy = true;
      pRegOld->nR = i->second.nR;
      pRegOld->fldtype = QDB_LEGACY::FLD_TYPE(i->second.fldtype);
      if (strlen(i->second.filename.c_str()) >= QDB_LEGACY::MAX_LEN_FILE_NAME) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      strncpy(pRegOld->filename, i->second.filename.c_str(), QDB_LEGACY::MAX_LEN_FILE_NAME);
      if (strlen(i->second.tbl.c_str()) >= QDB_LEGACY::MAX_LEN_TBL_NAME) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      strncpy(pRegOld->tbl, i->second.tbl.c_str(), QDB_LEGACY::MAX_LEN_TBL_NAME);
      if (strlen(i->second.h_fld.c_str()) >= QDB_LEGACY::MAX_LEN_FLD_NAME) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      strncpy(pRegOld->h_fld, i->second.h_fld.c_str(), QDB_LEGACY::MAX_LEN_FLD_NAME);
      if (strlen(i->second.d_fld.c_str()) >= QDB_LEGACY::MAX_LEN_FLD_NAME) {
        whereami(__FILE__, __LINE__);
        return Q_FAIL;
      }
      strncpy(pRegOld->d_fld, i->second.d_fld.c_str(), QDB_LEGACY::MAX_LEN_FLD_NAME);
    }
    return  Q_PASS;
  }
#endif // SUPPORT_LEGACY
}
